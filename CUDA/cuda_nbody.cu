
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <iomanip>
#include <chrono>

using namespace std;

// Constants
const double G = 6.67430e-11;  // Gravitational constant
const double TIME_STEP = 0.01; // Time step for simulation
const int NUM_BODIES = 1000;   // Number of bodies
const int STEPS = 10000;       // Simulation steps
const int BLOCK_SIZE = 256;    // CUDA thread block size

// Structure to represent a body (planet, particle)
struct Body {
    double x, y, vx, vy, mass;
};

// Structure for aligned force data
struct Force {
    double fx;
    double fy;
};

// CUDA kernel for computing forces
__global__ void computeForcesKernel(const Body* bodies, Force* forces, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n) {
        double fx = 0.0;
        double fy = 0.0;

        // Compute forces with all other bodies
        for (int j = 0; j < n; j++) {
            if (idx != j) {
                double dx = bodies[j].x - bodies[idx].x;
                double dy = bodies[j].y - bodies[idx].y;
                double dist = sqrt(dx * dx + dy * dy + 1e-9);
                double force = (G * bodies[idx].mass * bodies[j].mass) / (dist * dist);

                fx += force * dx / dist;
                fy += force * dy / dist;
            }
        }

        forces[idx].fx = fx;
        forces[idx].fy = fy;
    }
}

// CUDA kernel for updating positions
__global__ void updateBodiesKernel(Body* bodies, const Force* forces, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n) {
        bodies[idx].vx += (forces[idx].fx / bodies[idx].mass) * TIME_STEP;
        bodies[idx].vy += (forces[idx].fy / bodies[idx].mass) * TIME_STEP;
        bodies[idx].x += bodies[idx].vx * TIME_STEP;
        bodies[idx].y += bodies[idx].vy * TIME_STEP;
    }
}

// Function to initialize bodies randomly
void initializeBodies(vector<Body>& bodies) {
    srand(time(0));
    for (int i = 0; i < bodies.size(); i++) {
        bodies[i] = {
            (double)(rand() % 1000),
            (double)(rand() % 1000),
            0.0, 0.0,
            (double)(rand() % 100 + 1)
        };
    }
}

int main() {
    vector<Body> bodies(NUM_BODIES);
    vector<Force> forces(NUM_BODIES);

    // Initialize bodies on host
    initializeBodies(bodies);

    // Allocate memory on device
    Body* d_bodies;
    Force* d_forces;
    hipMalloc(&d_bodies, NUM_BODIES * sizeof(Body));
    hipMalloc(&d_forces, NUM_BODIES * sizeof(Force));

    // Copy initial data to device
    hipMemcpy(d_bodies, bodies.data(), NUM_BODIES * sizeof(Body), hipMemcpyHostToDevice);

    // Calculate grid dimensions
    int numBlocks = (NUM_BODIES + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Start timing
    auto start_time = chrono::high_resolution_clock::now();

    // Main simulation loop
    for (int step = 0; step < STEPS; step++) {
        // Compute forces
        computeForcesKernel<<<numBlocks, BLOCK_SIZE>>>(d_bodies, d_forces, NUM_BODIES);
        
        // Update positions
        updateBodiesKernel<<<numBlocks, BLOCK_SIZE>>>(d_bodies, d_forces, NUM_BODIES);
        
        // Synchronize to ensure step completion
        hipDeviceSynchronize();
    }

    // Copy final results back to host
    hipMemcpy(bodies.data(), d_bodies, NUM_BODIES * sizeof(Body), hipMemcpyDeviceToHost);

    // Calculate final statistics
    double total_kinetic_energy = 0.0;
    double total_distance = 0.0;

    for (const auto& body : bodies) {
        double velocity = sqrt(body.vx * body.vx + body.vy * body.vy);
        total_kinetic_energy += 0.5 * body.mass * velocity * velocity;
        total_distance += sqrt(body.x * body.x + body.y * body.y);
    }

    auto end_time = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(end_time - start_time);

    // Print statistics in a consistent format
    cout << "=== Simulation Statistics ===\n";
    cout << "Implementation: CUDA\n";
    cout << "Bodies: " << NUM_BODIES << "\n";
    cout << "Steps: " << STEPS << "\n";
    cout << "Block Size: " << BLOCK_SIZE << "\n";
    cout << "Blocks: " << numBlocks << "\n";
    cout << "Execution Time: " << duration.count() << "\n";
    cout << "Kinetic Energy: " << scientific << setprecision(3) << total_kinetic_energy << "\n";
    cout << "Avg Distance: " << fixed << setprecision(2) << total_distance/NUM_BODIES << "\n";
    cout << "=== End Statistics ===\n";

    // Clean up
    hipFree(d_bodies);
    hipFree(d_forces);
    return 0;
}
